
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>

// cuda 中 host 表示 cpu 端，device 表示 gpu 端
// __device__ 是设备函数的声明符号，表明该函数在 device 执行，且只能在 device
// 中调用
__device__ const char *device_hello_world(void) {
  return "GPU: Hello world!\n";
}

// __host__ 是主机函数的声明符号，表明该函数在 host 执行，且只能在 host 中调用
__host__ const char *host_hello_world(void) { return "CPU: Hello world!\n"; }

// __global__ 是核函数的声明符号，表明该函数在 device 执行，且只能在 host 中调用
__global__ void hello_world(void) {
  const char *str = device_hello_world();
  printf("%s", str);
}

int main(int argc, char **argv) {
  printf("%s", host_hello_world());
  // <<<grid_dim, block_dim>>> 是核函数的调用符号，表示启动 grid_dim 个 block，
  // 每个 block 有 block_dim 个线程
  hello_world<<<1, 10>>>();
  hipDeviceReset();
  return 0;
}
